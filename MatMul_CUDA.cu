
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include <iostream>
#include <stdlib.h>
#include <sys/time.h>

#define ROWS 2
#define COLS 2

using namespace std;

__global__ void matrix_mul_gpu(float* M, float* N, float* P, int width)
{
    for(int i=0;i<width;i++)
        for(int j=0;j<width;j++)
        {
            float sum = 0.0;
            for(int k=0;k<width;k++)
            {
                float a = M[i*width+k];
                float b = N[k*width+j];
                sum += a*b;
            }
            P[i*width+j] = sum;
        }
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    float *A, *B, *C;
    int total_size = ROWS*COLS*sizeof(float);
    A = (float*)malloc(total_size);
    B = (float*)malloc(total_size);
    C = (float*)malloc(total_size);

    //CPU一维数组初始化
    for(int i=0;i<ROWS*COLS;i++)
    {
        A[i] = 2.0;
        B[i] = 1.0;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, total_size);
    hipMalloc((void **)&d_B, total_size);
    hipMalloc((void **)&d_C, total_size);

    hipMemcpy(d_A, A, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, total_size, hipMemcpyHostToDevice);


    dim3 blockSize(4);  //一个block包含4个线程
    dim3 gridSize((4 + blockSize.x - 1) / blockSize.x);  //(1048576+256-1)/256=4096

    // 执行kernel
    matrix_mul_gpu<<<gridSize, blockSize>>>(d_A,d_B,d_C,COLS);

        // 将device得到的结果拷贝到host
    hipMemcpy(C, d_C, total_size, hipMemcpyDeviceToHost);  //d_z拷贝到z

    // 检查执行结果
    for (int i = 0; i < COLS; i++)
    {
        for (int j=0;j<COLS;j++)
            std::cout << C[i*COLS+j] << std::endl;
    }
    	
    // 释放device内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放host内存
    free(A);
    free(B);
    free(C);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;

    return 0;
}


