
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>


__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = index+5; i < n; i++)
    {		
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 10;
    int nBytes = N * sizeof(float);

    // 申请host内存
    // malloc 返回是void*，无法直接赋值给float型指针，于是采用(float*)强制转换
    float *x, *y, *z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    // 初始化数据
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    // 申请device内存
    float *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, nBytes);
    hipMalloc((void **)&d_y, nBytes);
    hipMalloc((void **)&d_z, nBytes);

    // 将host数据拷贝到device
    hipMemcpy(d_x, x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, nBytes, hipMemcpyHostToDevice);

    // 定义kernel的执行配置
    dim3 blockSize(4);  //一个block包含4个线程
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);  //(1048576+256-1)/256=4096

    // 执行kernel
    add  <<< gridSize, blockSize >>> (d_x, d_y, d_z, N);

    // 将device得到的结果拷贝到host
    hipMemcpy(z, d_z, nBytes, hipMemcpyDeviceToHost);  //d_z拷贝到z

    // 检查执行结果
    for (int i = 0; i < N; i++)
    	std::cout << z[i] << std::endl;


    // 释放device内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    // 释放host内存
    free(x);
    free(y);
    free(z);

    return 0;
}


