
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>


int main()
{
    int dev;
    hipGetDeviceCount(&dev);
    std::cout<<dev<<std::endl;
    for (int i = 0; i < dev; i++) 
    {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);
	std::cout << "device name:" << i << ": " << prop.name << std::endl;
    	std::cout << "SM的数量：" << prop.multiProcessorCount << std::endl;
    	std::cout << "每个block线程块的共享内存大小：" << prop.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    	std::cout << "每个block线程块的最大线程数：" << prop.maxThreadsPerBlock << std::endl;
    	std::cout << "每个SM的最大线程数：" << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "warp size:：" << prop.warpSize << std::endl;
    	std::cout << "num of warp(SP) per SM：" << prop.maxThreadsPerMultiProcessor / 32 << std::endl;
        return 0;
    }
}


